
#include <hip/hip_runtime.h>
#include <iostream>

int main(int argc, char* argv[])
{
    hipDeviceProp_t dev_prop;
    int dev_cnt = 0;
    hipGetDeviceCount(&dev_cnt);
    for(int i=0; i < dev_cnt; ++i)
    {
        hipGetDeviceProperties(&dev_prop, i);
        std::cout << "Device : " << i << " has compute capability " << dev_prop.major << "." << dev_prop.minor << std::endl;
    }
    return 0;
}
