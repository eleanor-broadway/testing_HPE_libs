#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <numeric>
#include <string>
#include <fstream>
#include <cassert>
#include <vector>
#include <cmath>
#include "timer.hpp"

typedef float value_type;
typedef std::size_t size_type;

static const int moment_block_x    = 256;
static const int diffusion_block_x = 16;
static const int diffusion_block_y = 16;

__global__ void diffusion_kernel(value_type * rho_out, value_type const * rho, value_type fac, int N)
{
    __shared__ value_type rho_loc[(diffusion_block_x+2)*(diffusion_block_y+2)];
    int const gj = blockIdx.x*blockDim.x + threadIdx.x;
    int const gi = blockIdx.y*blockDim.y + threadIdx.y;

    int const lN = diffusion_block_y+2;
    int const lj = threadIdx.x + 1;
    int const li = threadIdx.y + 1;

    if(gi < N && gj < N)
    {
        // Load the bulk
        rho_loc[li*lN + lj] = rho[gi*N + gj];

        // Load the ghost cells
        if(threadIdx.y == 0)
        {
            rho_loc[(li-1)*lN + lj] = (gi == 0 ? 0 : rho[(gi-1)*N + gj]);
        }
        if(threadIdx.y == blockDim.y-1)
        {
            rho_loc[(lN-1)*lN + lj] = (gi == N-1 ? 0 : rho[(gi+1)*N + gj]);
        }
        if(threadIdx.x == 0)
        {
            rho_loc[li*lN + lj-1] = (gj == 0 ? 0 : rho[gi*N + gj-1]);
        }
        if(threadIdx.x == blockDim.x-1)
        {
            rho_loc[li*lN + lN-1] = (gj == N-1 ? 0 : rho[gi*N + gj+1]);
        }
    }
    __syncthreads();

    if(gi < N && gj < N)
    {
        rho_out[gi*N + gj] = rho_loc[li*lN + lj] + fac
            *
            (  rho_loc[li*lN + (lj+1)]
             + rho_loc[li*lN + (lj-1)]
             + rho_loc[(li+1)*lN + lj]
             + rho_loc[(li-1)*lN + lj]
             - 4*rho_loc[li*lN + lj]
             );
    }
}

__global__ void get_moment_kernel(value_type * result, value_type const * rho, value_type rmin, value_type dr, int N)
{
    __shared__ value_type partial_sums[256];
    partial_sums[threadIdx.x] = 0;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(j < N)
    {
        value_type x = j*dr + rmin;

        // Note: One could improve this loop by replacing it with another
        // parallel reduction along this direction. We do not bother,
        // because this kernel is not the bottleneck of our simulation.
        for(int i=0; i < N; ++i)
        {
            value_type y = i*dr + rmin;
            partial_sums[threadIdx.x] += rho[i*N+j] * (x*x + y*y);
            // The index i (instead of j) of this loop may seem funny because
            // it jumps in strides of N. However, contrary to locality the CPU,
            // locality on the GPU means neighboring threads (threadIdx.x)
            // access neighboring memory locations.
        }
    }
    __syncthreads();
    // Simple reduction
    if(threadIdx.x < 128)
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x+128];
    __syncthreads();
    if(threadIdx.x < 64)
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x+64];
    __syncthreads();
    if(threadIdx.x < 32)
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x+32];
    __syncthreads();
    if(threadIdx.x < 16)
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x+16];
    __syncthreads();
    if(threadIdx.x < 8)
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x+8];
    __syncthreads();
    if(threadIdx.x < 4)
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x+4];
    __syncthreads();
    if(threadIdx.x < 2)
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x+2];
    __syncthreads();
    if(threadIdx.x < 1)
    {
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x+1];
        result[blockIdx.x] = dr*dr*partial_sums[threadIdx.x];
    }
}

class Diffusion2D
{
    
public:
    
    Diffusion2D(
                const value_type D,
                const value_type rmax,
                const value_type rmin,
                const size_type N
                )
    : D_(D)
    , rmax_(rmax)
    , rmin_(rmin)
    , N_(N)
    , N_tot(N*N)
    , d_rho_(0)
    , d_rho_tmp_(0)
    {
        N_tot = N_*N_;
        
        /// real space grid spacing
        dr_ = (rmax_ - rmin_) / (N_ - 1);
        
        /// dt < dx*dx / (4*D) for stability
        dt_ = dr_ * dr_ / (6 * D_);
        
        /// stencil factor
        fac_ = dt_ * D_ / (dr_ * dr_);

        // Allocate memory on Device
        hipMalloc(&d_rho_, N_tot*sizeof(value_type));
        hipMalloc(&d_rho_tmp_, N_tot*sizeof(value_type));
        hipMalloc(&d_moment_, (N/moment_block_x) * sizeof(value_type) );
        
        hipMemset(d_rho_,0,N_tot);
        hipMemset(d_rho_tmp_,0,N_tot);
        
        InitializeSystem();
    }
    
    ~Diffusion2D()
    {
        hipFree(d_moment_);
        hipFree(d_rho_tmp_);
        hipFree(d_rho_);
    }
    
    void PropagateDensity(int steps);
    
    value_type GetMoment() const {
        int const blocks = (N_+moment_block_x-1)/moment_block_x;
        std::vector<value_type> moment(blocks);
        get_moment_kernel<<<blocks,moment_block_x>>>(d_moment_, d_rho_, rmin_, dr_, N_);
        hipMemcpy(&moment[0], d_moment_, blocks * sizeof(value_type), hipMemcpyDeviceToHost);

        return std::accumulate(moment.begin(),moment.end(),0.0);
    }
    
    value_type GetTime() const {return time_;}
    
    void WriteDensity(const std::string file_name) const;
    
private:
    
    void InitializeSystem();
    
    const value_type D_, rmax_, rmin_;
    const size_type N_;
    size_type N_tot;
    
    value_type dr_, dt_, fac_;
    
    value_type time_;
    
    value_type *d_rho_, *d_rho_tmp_;
    value_type *d_moment_;
};

void Diffusion2D::WriteDensity(const std::string file_name) const
{
    // Get data from device
    std::vector<value_type> rho(N_*N_);
    hipMemcpy(&rho[0], d_rho_, rho.size() * sizeof(value_type), hipMemcpyDeviceToHost);

    std::ofstream out_file;
    out_file.open(file_name.c_str(), std::ios::out);
    if(out_file.good())
    {
        for(size_type i = 0; i < N_; ++i){
            for(size_type j = 0; j < N_; ++j)
                out_file << (i*dr_+rmin_) << '\t' << (j*dr_+rmin_) << '\t' << rho[i*N_ + j] << "\n";
            
            out_file << "\n";
        }
    }
    
    out_file.close();
}

void Diffusion2D::PropagateDensity(int steps)
{
    using std::swap;
    /// Dirichlet boundaries; central differences in space, forward Euler
    /// in time

    dim3 block_size(diffusion_block_x,diffusion_block_y,1);
    dim3 grid_size((N_+diffusion_block_x-1)/diffusion_block_x,(N_+diffusion_block_y-1)/diffusion_block_y,1); // Round-up needed number of blocks (N/block_size)
    for(int s = 0; s < steps; ++s)
    {
        diffusion_kernel<<<grid_size, block_size>>>(d_rho_tmp_, d_rho_, fac_, N_);
        swap(d_rho_, d_rho_tmp_);
        time_ += dt_;
    }
}

void Diffusion2D::InitializeSystem()
{
    std::vector<value_type> rho(N_*N_);
    time_ = 0;
    
    /// initialize rho(x,y,t=0)
    value_type bound = 1./2;
    
    for(size_type i = 0; i < N_; ++i){
        for(size_type j = 0; j < N_; ++j){
            if(std::fabs(i*dr_+rmin_) < bound && std::fabs(j*dr_+rmin_) < bound){
                rho[i*N_ + j] = 1;
            }
            else{
                rho[i*N_ + j] = 0;
            }
            
        }
    }
    hipMemcpy(d_rho_, &rho[0], rho.size() * sizeof(value_type), hipMemcpyHostToDevice);
}

int main(int argc, char* argv[])
{
    if(argc != 2)
    {
        std::cerr << "usage: " << argv[0] << " <log2(size)>" << std::endl;
        return 1;
    }

    const value_type D = 1;
    const value_type tmax = 0.01;
    const value_type rmax = 1;
    const value_type rmin = -1;
    
    const size_type N_ = 1 << std::atoi(argv[1]);
    const int steps_between_measurements = 100;
    
    Diffusion2D System(D, rmax, rmin, N_);
    
    value_type time = 0;
    
    timer runtime;
    runtime.start();
    
    while(time < tmax){
        System.PropagateDensity(steps_between_measurements);
        time = System.GetTime();
        value_type moment = System.GetMoment();
        std::cout << time << '\t' << moment << std::endl;
    }
    
    runtime.stop();
    
    double elapsed = runtime.get_timing();
    
    std::cerr << argv[0] << "\t N=" <<N_ << "\t time=" << elapsed << "s" << std::endl;
    
    std::string density_file = "Density.dat";
    System.WriteDensity(density_file);
    
    return 0;
}
