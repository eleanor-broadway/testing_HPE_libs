#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <numeric>
#include <string>
#include <fstream>
#include <cassert>
#include <vector>
#include <cmath>
#include "timer.hpp"

typedef float value_type;
typedef std::size_t size_type;

static const int diffusion_block_x = 16;
static const int diffusion_block_y = 16;

__global__ void diffusion_kernel(value_type * rho_out, value_type const * rho, value_type fac, int N)
{
    int const j = blockIdx.x*blockDim.x + threadIdx.x;
    int const i = blockIdx.y*blockDim.y + threadIdx.y;
    if(i < N && j < N)
    {
        rho_out[i*N + j] = rho[i*N + j] + fac
            *
            (
             (j == N-1 ? 0 : rho[i*N + (j+1)])
             +
             (j == 0 ? 0 : rho[i*N + (j-1)])
             +
             (i == N-1 ? 0 : rho[(i+1)*N + j])
             +
             (i == 0 ? 0 : rho[(i-1)*N + j])
             -
             4*rho[i*N + j]
             );
    }
}

class Diffusion2D
{
    
public:
    
    Diffusion2D(
                const value_type D,
                const value_type rmax,
                const value_type rmin,
                const size_type N
                )
    : D_(D)
    , rmax_(rmax)
    , rmin_(rmin)
    , N_(N)
    , N_tot(N*N)
    , d_rho_(0)
    , d_rho_tmp_(0)
    , rho_(N_tot)
    {
        /// real space grid spacing
        dr_ = (rmax_ - rmin_) / (N_ - 1);
        
        /// dt < dx*dx / (4*D) for stability
        dt_ = dr_ * dr_ / (6 * D_);
        
        /// stencil factor
        fac_ = dt_ * D_ / (dr_ * dr_);

        // Allocate memory on Device
        hipMalloc(&d_rho_, N_tot*sizeof(value_type));
        hipMalloc(&d_rho_tmp_, N_tot*sizeof(value_type));
        
        hipMemset(d_rho_,0,N_tot);
        hipMemset(d_rho_tmp_,0,N_tot);
        
        InitializeSystem();
    }
    
    ~Diffusion2D()
    {
        hipFree(d_rho_tmp_);
        hipFree(d_rho_);
    }
    
    void PropagateDensity(int steps);
    
    value_type GetMoment() {
        hipMemcpy(&rho_[0], d_rho_, rho_.size() * sizeof(value_type), hipMemcpyDeviceToHost);
        value_type sum = 0;
        
        for(size_type i = 0; i < N_; ++i)
            for(size_type j = 0; j < N_; ++j) {
                value_type x = j*dr_ + rmin_;
                value_type y = i*dr_ + rmin_;
                sum += rho_[i*N_ + j] * (x*x + y*y);
            }
        
        return dr_*dr_*sum;
    }
    
    value_type GetTime() const {return time_;}
    
    void WriteDensity(const std::string file_name) const;
    
private:
    
    void InitializeSystem();
    
    const value_type D_, rmax_, rmin_;
    const size_type N_;
    size_type N_tot;
    
    value_type dr_, dt_, fac_;
    
    value_type time_;
    
    value_type *d_rho_, *d_rho_tmp_;
    mutable std::vector<value_type> rho_;
};

void Diffusion2D::WriteDensity(const std::string file_name) const
{
    // Get data from device
    hipMemcpy(&rho_[0], d_rho_, rho_.size() * sizeof(value_type), hipMemcpyDeviceToHost);
    std::ofstream out_file;
    out_file.open(file_name.c_str(), std::ios::out);
    if(out_file.good())
    {
        for(size_type i = 0; i < N_; ++i){
            for(size_type j = 0; j < N_; ++j)
                out_file << (i*dr_+rmin_) << '\t' << (j*dr_+rmin_) << '\t' << rho_[i*N_ + j] << "\n";
            
            out_file << "\n";
        }
    }
    
    out_file.close();
}

void Diffusion2D::PropagateDensity(int steps)
{
    using std::swap;
    /// Dirichlet boundaries; central differences in space, forward Euler
    /// in time

    dim3 block_size(diffusion_block_x,diffusion_block_y,1);
    dim3 grid_size((N_+diffusion_block_x-1)/diffusion_block_x,(N_+diffusion_block_y-1)/diffusion_block_y,1); // Round-up needed number of blocks (N/block_size)
    for(int s = 0; s < steps; ++s)
    {
        diffusion_kernel<<<grid_size, block_size>>>(d_rho_tmp_, d_rho_, fac_, N_);
        swap(d_rho_, d_rho_tmp_);
        time_ += dt_;
    }
}

void Diffusion2D::InitializeSystem()
{
    time_ = 0;
    
    /// initialize rho(x,y,t=0)
    value_type bound = 1./2;
    
    for(size_type i = 0; i < N_; ++i){
        for(size_type j = 0; j < N_; ++j){
            if(std::fabs(i*dr_+rmin_) < bound && std::fabs(j*dr_+rmin_) < bound){
                rho_[i*N_ + j] = 1;
            }
            else{
                rho_[i*N_ + j] = 0;
            }
            
        }
    }
    hipMemcpy(d_rho_, &rho_[0], rho_.size() * sizeof(value_type), hipMemcpyHostToDevice);
}

int main(int argc, char* argv[])
{
    if(argc != 2)
    {
        std::cerr << "usage: " << argv[0] << " <log2(size)>" << std::endl;
        return 1;
    }

    const value_type D = 1;
    const value_type tmax = 0.01;
    const value_type rmax = 1;
    const value_type rmin = -1;
    
    const size_type N_ = 1 << std::atoi(argv[1]);
    const int steps_between_measurements = 100;
    
    Diffusion2D System(D, rmax, rmin, N_);
    
    value_type time = 0;
    
    timer runtime;
    runtime.start();
    
    while(time < tmax){
        System.PropagateDensity(steps_between_measurements);
        time = System.GetTime();
        value_type moment = System.GetMoment();
        std::cout << time << '\t' << moment << std::endl;
    }
    
    runtime.stop();
    
    double elapsed = runtime.get_timing();
    
    std::cerr << argv[0] << "\t N=" <<N_ << "\t time=" << elapsed << "s" << std::endl;
    
    std::string density_file = "Density.dat";
    System.WriteDensity(density_file);
    
    return 0;
}
