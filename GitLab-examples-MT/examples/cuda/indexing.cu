/*
 *  HPC Class 
 *  GPU Examples of Indexes and Indexing
 *
 */


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define N 20


// GPU kernels:
__global__ void kernel1(int* a)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	a[idx] = 7;
}

__global__ void kernel2(int* b)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	b[idx] = blockIdx.x;
}

__global__ void kernel3(int* c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	c[idx] = threadIdx.x;
}


int main()
{
	int h_a[N], h_b[N],h_c[N];  // h stands for host (stuffs on CPU)
	int* d_pa, *d_pb, *d_pc;	// d stands for device (stuffs on GPU)	
	
	//allocate the memory on the GPU
	hipMalloc( (void**)&d_pa, N*sizeof(int) );
	hipMalloc( (void**)&d_pb, N*sizeof(int) );
	hipMalloc( (void**)&d_pc, N*sizeof(int) );
	
	hipMemset(d_pa, 0, N);
	hipMemset(d_pb, 0, N);
	hipMemset(d_pc, 0, N);
	
	// call the GPU kernels
	kernel1<<<5,4>>>(d_pa);
	kernel2<<<5,4>>>(d_pb);
	kernel3<<<5,4>>>(d_pc);
	
	// copy the arrays back from the GPU to the CPU	
	hipMemcpy(h_a, d_pa, N*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_pb, N*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_c, d_pc, N*sizeof(int), hipMemcpyDeviceToHost);
	
	// display the results
	cout<< " Results from kernel1:" << endl;
	for (int i = 0; i<N; i++)
		cout<< h_a[i] << " ";
	cout<< endl;
	
	cout<< " Results from kernel2:" << endl;
	for (int i = 0; i<N; i++)
		cout<< h_b[i] << " ";
	cout<< endl;
	
	cout<< " Results from kernel3:" << endl;
	for (int i = 0; i<N; i++)
		cout<< h_c[i] << " ";
	cout<< endl;
	
	
	//free the memory allocated on the GPU
	hipFree(d_pa);
	hipFree(d_pb);
    hipFree(d_pc);
	
	return 0;
	
} 